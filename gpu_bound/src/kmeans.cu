#include "hip/hip_runtime.h"
#include "kmeans.h"

#define NUM_BLOCKS 78128
#define NUM_THREADS_PER_BLOCK 128
#define N NUM_BLOCKS*NUM_THREADS_PER_BLOCK // + x if wanted

#define K 32

using namespace std;



typedef struct Point
{
    float x;
    float y;
} Point;

typedef struct ClustersInfo
{
    float * sum_points_x;
    float * sum_points_y;
    int * sizes;
} ClustersInfo;

typedef struct Output
{
    Point * clusters_center;
    int * clusters_size;
    int iterations;
} Output;


void init(Point * sample, Point * clusters_center)
{
    srand(10);

    for (int i = 0; i < N; i++)
    {
        float x = (float) rand() / RAND_MAX;
        float y = (float) rand() / RAND_MAX;

        Point point = 
        {
            .x = x, 
            .y = y
        };

        sample[i] = point;
    }

    for (int i = 0; i < K; i++)
        clusters_center[i] = sample[i];
}

__device__
inline float get_sq_euclidean_dist(Point a, Point b)
{
    float dx = (a.x - b.x);
    float dy = (a.y - b.y);

    return dx * dx + dy * dy;
}


__global__
void kmeansKernel (Point * sample, Point * global_clusters_center, ClustersInfo clusters_info) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int lid = threadIdx.x; // local thread id within a block


	__shared__ Point clusters_center[K];
    __shared__ float sum_points_x[K];
    __shared__ float sum_points_y[K];
    __shared__ int sizes[K];

    if (id < N)
    {

        if (lid == 0)
        {
            for (int j = 0; j < K; j++)
            {
                clusters_center[j] = global_clusters_center[j];
                sum_points_x[j] = 0.0f;
                sum_points_y[j] = 0.0f;
                sizes[j] = 0;
            }
        } 
        __syncthreads(); // wait for all threads within a block


        Point point = sample[id];

        int best_cluster = 0;

        float best_cluster_dist = get_sq_euclidean_dist(clusters_center[0], point);
        

        for (int j = 1; j < K; j++)
        {
            float cluster_dist = get_sq_euclidean_dist(clusters_center[j], point);

            if (cluster_dist < best_cluster_dist)
            {
                best_cluster = j;
                best_cluster_dist = cluster_dist;
            }
        }

        /*
        ARCH > 6.0
        atomicAdd_block(&sum_points_x[best_cluster], point.x);
        atomicAdd_block(&sum_points_y[best_cluster], point.y);
        atomicAdd_block(&sizes[best_cluster], 1);
        */


        atomicAdd(&sum_points_x[best_cluster], point.x);
        atomicAdd(&sum_points_y[best_cluster], point.y);
        atomicAdd(&sizes[best_cluster], 1);


        __syncthreads(); // wait for all threads within a block
        if (lid == 0)
        {
            for (int j = 0; j < K; j++)
            {
                atomicAdd(&clusters_info.sum_points_x[j], sum_points_x[j]);
                atomicAdd(&clusters_info.sum_points_y[j], sum_points_y[j]);
                atomicAdd(&clusters_info.sizes[j], sizes[j]);
            }
        }
    }
}


Point * reevaluate_centers(ClustersInfo clusters_info)
{
    Point * new_clusters_center = (Point *) malloc(K * sizeof(Point));

    for (int i = 0; i < K; i++)
        new_clusters_center[i] = (Point) {
            .x = clusters_info.sum_points_x[i] / clusters_info.sizes[i],
            .y = clusters_info.sum_points_y[i] / clusters_info.sizes[i],
        };


    return new_clusters_center;
}


int has_converged(Point * clusters_center, Point * new_clusters_center)
{
    return memcmp(clusters_center, new_clusters_center, K * sizeof(Point)) == 0 ? 1 : 0;
}


void initKmeansKernel(Point * sample, Point ** _device_sample, Point ** _device_clusters_center, ClustersInfo * _device_clusters_info)
{

    // declare variable with size of the array in bytes
	int sample_bytes = N * sizeof(Point);
    int clusters_bytes = K * sizeof(Point);
	int clusters_coord_bytes = K * sizeof(float);
	int clusters_sizes_bytes = K * sizeof(int);

    // pointers to the device memory
	Point * device_sample;
	Point * device_clusters_center;
	ClustersInfo device_clusters_info;

    // allocate the memory on the device
	hipMalloc((void**) &device_sample, sample_bytes);
	hipMalloc((void**) &device_clusters_center, clusters_bytes);
	hipMalloc((void**) &device_clusters_info.sum_points_x, clusters_coord_bytes);
    hipMalloc((void**) &device_clusters_info.sum_points_y, clusters_coord_bytes);
	hipMalloc((void**) &device_clusters_info.sizes, clusters_sizes_bytes);
	checkCUDAError("mem allocation");

    // copy inputs to the device
	hipMemcpy(device_sample, sample, sample_bytes, hipMemcpyHostToDevice);
    checkCUDAError("memcpy h->d");

    // Return Values
    *_device_sample = device_sample;
	*_device_clusters_center = device_clusters_center;
	*_device_clusters_info = device_clusters_info;
}

void freeKmeansKernel(Point * device_sample, Point * device_clusters_center, ClustersInfo device_clusters_info)
{
    // free the device memory
	hipFree(device_sample);
	hipFree(device_clusters_center);
	hipFree(device_clusters_info.sum_points_x);
    hipFree(device_clusters_info.sum_points_y);
	hipFree(device_clusters_info.sizes);
	checkCUDAError("mem free");
}


void launchKmeansKernel(Point * clusters_center, ClustersInfo clusters_info,
                        Point * device_sample, Point * device_clusters_center, ClustersInfo device_clusters_info)
{
	// declare variable with size of the array in bytes
    int clusters_bytes = K * sizeof(Point);
	int clusters_coord_bytes = K * sizeof(float);
	int clusters_sizes_bytes = K * sizeof(int);

    // reset inputs to the device
    hipMemset(device_clusters_info.sum_points_x, 0.0f, clusters_coord_bytes);
    hipMemset(device_clusters_info.sum_points_y, 0.0f, clusters_coord_bytes);
    hipMemset(device_clusters_info.sizes, 0, clusters_sizes_bytes);

	// copy inputs to the device
	hipMemcpy(device_clusters_center, clusters_center, clusters_bytes, hipMemcpyHostToDevice);
	checkCUDAError("memcpy h->d");

	// launch the kernel
	//startKernelTime();
	kmeansKernel <<< NUM_BLOCKS, NUM_THREADS_PER_BLOCK >>> (device_sample, device_clusters_center, device_clusters_info);
	//stopKernelTime();
	checkCUDAError("kernel invocation");

	// copy the output to the host
	hipMemcpy(clusters_info.sum_points_x, device_clusters_info.sum_points_x, clusters_coord_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(clusters_info.sum_points_y, device_clusters_info.sum_points_y, clusters_coord_bytes, hipMemcpyDeviceToHost);
	hipMemcpy(clusters_info.sizes, device_clusters_info.sizes, clusters_sizes_bytes, hipMemcpyDeviceToHost);
	checkCUDAError("memcpy d->h");
}

Output find_centers(Point * sample, Point * clusters_center)
{
    int finished;
    int iterations = 0;

    float * sum_points_x = (float *) malloc(K * sizeof(float));
    float * sum_points_y = (float *) malloc(K * sizeof(float));
    int * clusters_size = (int *) malloc(K * sizeof(int));

    ClustersInfo clusters_info =
    {
        .sum_points_x = sum_points_x,
        .sum_points_y = sum_points_y,
        .sizes = clusters_size
    };


    // pointers to the device memory
	Point * device_sample;
	Point * device_clusters_center;
	ClustersInfo device_clusters_info;

    initKmeansKernel(sample, &device_sample, &device_clusters_center, &device_clusters_info);


    do {
		launchKmeansKernel(clusters_center, clusters_info, device_sample, device_clusters_center, device_clusters_info);

        Point * new_clusters_center = reevaluate_centers(clusters_info);

        finished = has_converged(clusters_center, new_clusters_center);

        free(clusters_center);
        clusters_center = new_clusters_center;
        ++iterations;

    } while(!finished && iterations < 21);

    iterations--; // Last iteration doesn't count because it's a verification

    free(clusters_info.sum_points_x);
    free(clusters_info.sum_points_y);
    freeKmeansKernel(device_sample, device_clusters_center, device_clusters_info);

    Output output = {
        .clusters_center = clusters_center,
        .clusters_size = clusters_size,
        .iterations = iterations
    };

    return output;
}

int main(void)
{
	Point * sample = (Point *) malloc(N * sizeof(Point));
    Point * clusters_center = (Point *) malloc(K * sizeof(Point));

    init(sample, clusters_center);

	Output output = find_centers(sample, clusters_center);

    printf("N = %d, K = %d\n", N, K);

    for (int i = 0; i < K; i++)
    {
        printf("Center: (%.3f, %.3f) : Size: %d\n", 
            output.clusters_center[i].x,
            output.clusters_center[i].y,
            output.clusters_size[i]
        );
    }

    printf("Iterations: %d\n", output.iterations);

    free(output.clusters_center);
    free(output.clusters_size);
    

    return 0;
}
